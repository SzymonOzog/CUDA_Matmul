#include "hip/hip_runtime.h"
#include "kernel_classes.cuh"
#include <cuda_pipeline.h>

template<int SM_TILES, int OUT_TILES>
__global__ void tensor_core_matmul_reg_smem_async(int n_elem, half* a, half* b, half* c)
{
    const int32_t warpM = (blockIdx.x*blockDim.x+threadIdx.x)/32;
    const int32_t warpN = blockIdx.y*blockDim.y+threadIdx.y;
    const int32_t laneM = threadIdx.x/32;
    const int32_t laneN = threadIdx.y;

    extern __shared__ char smem[];

    half (*a_smem)[SM_TILES][WMMA_MKN*WMMA_MKN]
        = reinterpret_cast<half(*)[SM_TILES][WMMA_MKN*WMMA_MKN]>(smem);
    half (*b_smem)[SM_TILES][WMMA_MKN*WMMA_MKN]
        = reinterpret_cast<half(*)[SM_TILES][WMMA_MKN*WMMA_MKN]>(
                smem + 2*SM_TILES*WMMA_MKN*WMMA_MKN*sizeof(half));

    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> a_frag[OUT_TILES];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_MKN, WMMA_MKN, WMMA_MKN, half> acc[OUT_TILES][OUT_TILES];

    for(int32_t i = 0; i<OUT_TILES; i++)
        for(int32_t j = 0; j<OUT_TILES; j++)
            nvcuda::wmma::fill_fragment(acc[i][j], 0);

    const int32_t matrix_a_row = warpM * WMMA_MKN * OUT_TILES;
    const int32_t matrix_b_col = warpN * WMMA_MKN * OUT_TILES;

    half* a_curr = a + blockIdx.x*SM_TILES*WMMA_MKN*n_elem;
    half* b_curr = b + blockIdx.y*SM_TILES*WMMA_MKN;
    for (int i = (threadIdx.y * blockDim.x + threadIdx.x)*8;
            i < SM_TILES*WMMA_MKN*WMMA_MKN;
            i+=blockDim.x*blockDim.y*8)
    {
        half* a_smem_curr = &a_smem[0][i/(WMMA_MKN*WMMA_MKN)][i%(WMMA_MKN*WMMA_MKN)];
        half* a_gmem_curr = &a_curr[(i/WMMA_MKN)*n_elem + i%WMMA_MKN];
        __pipeline_memcpy_async(a_smem_curr, a_gmem_curr, 16);

        half* b_smem_curr = &b_smem[0][(i/WMMA_MKN)%SM_TILES][(i/(SM_TILES*WMMA_MKN))*WMMA_MKN + i%(WMMA_MKN)];
        half* b_gmem_curr = &b_curr[(i/(SM_TILES*WMMA_MKN))*n_elem + i%(SM_TILES*WMMA_MKN)];
        __pipeline_memcpy_async(b_smem_curr, b_gmem_curr, 16);
    }
    __pipeline_commit();

    for (int32_t tile = 0; tile < n_elem; tile+=WMMA_MKN)
    {
        int stage = (tile/WMMA_MKN)%2;
        if (tile+WMMA_MKN<n_elem)
        {
            half* a_curr = a + blockIdx.x*SM_TILES*WMMA_MKN*n_elem + tile+WMMA_MKN;
            half* b_curr = b + (tile+WMMA_MKN)*n_elem + blockIdx.y*SM_TILES*WMMA_MKN;
            for (int i = (threadIdx.y * blockDim.x + threadIdx.x)*8;
                    i < SM_TILES*WMMA_MKN*WMMA_MKN;
                    i+=blockDim.x*blockDim.y*8)
            {
                int load_stage = (stage+1)%2;
                half* a_smem_curr = &a_smem[load_stage][i/(WMMA_MKN*WMMA_MKN)][i%(WMMA_MKN*WMMA_MKN)];
                half* a_gmem_curr = &a_curr[(i/WMMA_MKN)*n_elem + i%WMMA_MKN];
                __pipeline_memcpy_async(a_smem_curr, a_gmem_curr, 16);

                half* b_smem_curr = &b_smem[load_stage][(i/WMMA_MKN)%SM_TILES][(i/(SM_TILES*WMMA_MKN))*WMMA_MKN + i%(WMMA_MKN)];
                half* b_gmem_curr = &b_curr[(i/(SM_TILES*WMMA_MKN))*n_elem + i%(SM_TILES*WMMA_MKN)];
                __pipeline_memcpy_async(b_smem_curr, b_gmem_curr, 16);
            }
            __pipeline_commit();
        }
        __pipeline_wait_prior(0);
        //TODO Why do we need syncthreads here?
        __syncthreads();
        for (int n = 0; n < OUT_TILES; n++)
        {
            nvcuda::wmma::load_matrix_sync(a_frag[n], a_smem[stage][laneM*OUT_TILES + n], WMMA_MKN);
        }
        for (int n = 0; n < OUT_TILES; n++)
        {
            nvcuda::wmma::load_matrix_sync(b_frag, b_smem[stage][laneN*OUT_TILES + n], WMMA_MKN);
            for (int m = 0; m < OUT_TILES; m++)
            {
                nvcuda::wmma::mma_sync(acc[m][n], a_frag[m], b_frag, acc[m][n]);
            }
        }
        __syncthreads();
    }

    for(int32_t i = 0; i<OUT_TILES; i++)
    {
        int32_t output_row = matrix_a_row + i*WMMA_MKN;
        for(int32_t j = 0; j<OUT_TILES; j++)
        {
            int32_t output_col = matrix_b_col + j*WMMA_MKN;
            if (output_row < n_elem && output_col < n_elem)
            {
                nvcuda::wmma::store_matrix_sync(c + output_row * n_elem + output_col, acc[i][j], n_elem, nvcuda::wmma::mem_row_major);
            }
        }
    }
}

template<int SMEM_TILES, int OUT_TILES>
double check_configuration_async(half* a, half*b, half* output, int N)
{
    dim3 dimBlock(1,1,1);
    dim3 dimGrid(1,1,1);

    int num_warps_x = SMEM_TILES/OUT_TILES;
    int num_warps_y = SMEM_TILES/OUT_TILES;
    dimBlock.x = num_warps_x * 32;
    dimBlock.y = num_warps_y;

    dimGrid.x = std::ceil((float)N/(SMEM_TILES*WMMA_MKN));
    dimGrid.y = std::ceil((float)N/(SMEM_TILES*WMMA_MKN));
    unsigned int smem_size = 2*2*SMEM_TILES*WMMA_MKN*WMMA_MKN*sizeof(half);
    hipFuncSetAttribute(reinterpret_cast<const void*>(tensor_core_matmul_reg_smem_async<SMEM_TILES), OUT_TILES>, hipFuncAttributeMaxDynamicSharedMemorySize, smem_size);

    return measure_performance([&](){ tensor_core_matmul_reg_smem_async<SMEM_TILES, OUT_TILES><<<dimGrid, dimBlock, smem_size>>>(N, a, b, output); });
}

double TensorCoresAsyncKernel::run(half* a, half* b, half* cublas_ref, int N)
{
    double matmul_time = std::numeric_limits<double>::max();

    matmul_time = std::min(matmul_time, check_configuration_async<8, 2>(a, b, output, N));
    test_output(cublas_ref, N);

    // matmul_time = std::min(matmul_time, check_configuration_async<9, 3>(a, b, output, N));
    // test_output(cublas_ref, N);

    matmul_time = std::min(matmul_time, check_configuration_async<8, 4>(a, b, output, N));
    test_output(cublas_ref, N);

    return matmul_time;
}
