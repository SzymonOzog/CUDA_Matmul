#include "hip/hip_runtime.h"
#include "kernel_classes.cuh"
#include "ptx_helpers.cuh"

template<int SM_TILES, int OUT_TILES>
__global__ void tensor_core_matmul_reg_smem(int n_elem, half* a, half* b, half* c)
{
    const int32_t warpM = (blockIdx.x*blockDim.x+threadIdx.x)/32;
    const int32_t warpN = blockIdx.y*blockDim.y+threadIdx.y;
    const int32_t laneM = threadIdx.x/32;
    const int32_t laneN = threadIdx.y;
    // const int32_t lane_id = threadIdx.x%32;

    extern __shared__ char smem[];

    half (*a_smem)[WMMA_MKN*WMMA_MKN]
        = reinterpret_cast<half(*)[WMMA_MKN*WMMA_MKN]>(smem);
    half (*b_smem)[WMMA_MKN*WMMA_MKN]
        = reinterpret_cast<half(*)[WMMA_MKN*WMMA_MKN]>(
                smem + SM_TILES*WMMA_MKN*WMMA_MKN*sizeof(half));

    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> a_frag[OUT_TILES];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_MKN, WMMA_MKN, WMMA_MKN, half> acc[OUT_TILES][OUT_TILES];
    // mma_tile<16, 16> a_tile[OUT_TILES];
    // mma_tile<16, 16> b_tile;
    // mma_tile<16, 16> acc[OUT_TILES][OUT_TILES];

    for(int32_t i = 0; i<OUT_TILES; i++)
        for(int32_t j = 0; j<OUT_TILES; j++)
            nvcuda::wmma::fill_fragment(acc[i][j], 0);

    const int32_t matrix_a_row = warpM * WMMA_MKN * OUT_TILES;
    const int32_t matrix_b_col = warpN * WMMA_MKN * OUT_TILES;

    for (int32_t tile = 0; tile < n_elem; tile+=WMMA_MKN)
    {
        half* a_curr = a + blockIdx.x*SM_TILES*WMMA_MKN*n_elem + tile;
        half* b_curr = b + (tile)*n_elem + blockIdx.y*SM_TILES*WMMA_MKN;
        for (int i = (threadIdx.y * blockDim.x + threadIdx.x)*8;
                i < SM_TILES*WMMA_MKN*WMMA_MKN;
                i+=blockDim.x*blockDim.y*8)
        {
            half* a_smem_curr = &a_smem[i/(WMMA_MKN*WMMA_MKN)][i%(WMMA_MKN*WMMA_MKN)];
            half* a_gmem_curr = &a_curr[(i/WMMA_MKN)*n_elem + i%WMMA_MKN];
            reinterpret_cast<float4*>(a_smem_curr)[0]
                = reinterpret_cast<float4*>(a_gmem_curr)[0];

            half* b_smem_curr = &b_smem[(i/WMMA_MKN)%SM_TILES][(i/(SM_TILES*WMMA_MKN))*WMMA_MKN + i%(WMMA_MKN)];
            half* b_gmem_curr = &b_curr[(i/(SM_TILES*WMMA_MKN))*n_elem + i%(SM_TILES*WMMA_MKN)];
            reinterpret_cast<float4*>(b_smem_curr)[0]
                = reinterpret_cast<float4*>(b_gmem_curr)[0];
        }
        __syncthreads();
        for (int n = 0; n < OUT_TILES; n++)
        {
            nvcuda::wmma::load_matrix_sync(a_frag[n], a_smem[laneM*OUT_TILES + n], WMMA_MKN);
            // load_tile_a(a_tile[n], a_smem[laneM*OUT_TILES + n], WMMA_MKN, lane_id);
        }
        for (int n = 0; n < OUT_TILES; n++)
        {
            nvcuda::wmma::load_matrix_sync(b_frag, b_smem[laneN*OUT_TILES + n], WMMA_MKN);
            // load_tile_b(b_tile, b_smem[laneN*OUT_TILES + n], WMMA_MKN, lane_id);
            for (int m = 0; m < OUT_TILES; m++)
            {
                nvcuda::wmma::mma_sync(acc[m][n], a_frag[m], b_frag, acc[m][n]);
                // mma(a_tile[m], b_tile, acc[m][n]);
            }
        }
        __syncthreads();
    }

    for(int32_t i = 0; i<OUT_TILES; i++)
    {
        int32_t output_row = matrix_a_row + i*WMMA_MKN;
        for(int32_t j = 0; j<OUT_TILES; j++)
        {
            int32_t output_col = matrix_b_col + j*WMMA_MKN;
            if (output_row < n_elem && output_col < n_elem)
            {
                // for (int k = 0; k<4; k++)
                // {
                //     reinterpret_cast<half2*>(&c[(output_row + (lane_id>>2) + (k%2)*8)*n_elem + output_col + (k/2)*8])[lane_id%4]
                //         = acc[i][j].x[k];
                // }
               nvcuda::wmma::store_matrix_sync(c + output_row * n_elem + output_col, acc[i][j], n_elem, nvcuda::wmma::mem_row_major);
            }
        }
    }
}

template<int SMEM_TILES, int OUT_TILES>
double check_configuration_smem(half* a, half*b, half* output, int N)
{
    dim3 dimBlock(1,1,1);
    dim3 dimGrid(1,1,1);

    int num_warps_x = SMEM_TILES/OUT_TILES;
    int num_warps_y = SMEM_TILES/OUT_TILES;
    dimBlock.x = num_warps_x * 32;
    dimBlock.y = num_warps_y;

    dimGrid.x = std::ceil((float)N/(SMEM_TILES*WMMA_MKN));
    dimGrid.y = std::ceil((float)N/(SMEM_TILES*WMMA_MKN));
    unsigned int smem_size = 2*SMEM_TILES*WMMA_MKN*WMMA_MKN*sizeof(half);
    hipFuncSetAttribute(reinterpret_cast<const void*>(tensor_core_matmul_reg_smem<SMEM_TILES), OUT_TILES>, hipFuncAttributeMaxDynamicSharedMemorySize, smem_size);

    return measure_performance([&](){ tensor_core_matmul_reg_smem<SMEM_TILES, OUT_TILES><<<dimGrid, dimBlock, smem_size>>>(N, a, b, output); });
}

double TensorCoresSmemKernel::run(half* a, half* b, half* cublas_ref, int N)
{
    double matmul_time = std::numeric_limits<double>::max();

    matmul_time = std::min(matmul_time, check_configuration_smem<8, 2>(a, b, output, N));
    // debug_print(a, N, true); 
    // debug_print(b, N, true); 
    //
    // debug_print(cublas_ref, N, false); 
    // debug_print(output, N, true); 
    test_output(cublas_ref, N);

    // matmul_time = std::min(matmul_time, check_configuration_smem<9, 3>(a, b, output, N));
    // test_output(cublas_ref, N);

    matmul_time = std::min(matmul_time, check_configuration_smem<8, 4>(a, b, output, N));

    test_output(cublas_ref, N);
    return matmul_time;
}
