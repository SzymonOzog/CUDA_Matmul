
#include <hip/hip_runtime.h>
#include <cmath>
#include <iomanip>
#include <iostream>
#include <cassert>
#include <hipblas.h>
#include <mma.h>
#include <random>
#include <vector>

#define TILE_WIDTH 32
#define BENCH_STEPS 100
#define WARMUP_STEPS 25
#define TIMINGS 2
#define START 11

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define ASSERT(cond, msg, args...) assert((cond) || !fprintf(stderr, (msg "\n"), args))

using datatype = half;
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void debug_print(datatype* matrix, int N, bool device)
{
    datatype* host_ptr;
    if (device)
    {
        host_ptr = new datatype[N*N];
        hipMemcpy(host_ptr, matrix, N*N*sizeof(datatype), hipMemcpyDeviceToHost);
    }
    else
    {
        host_ptr = matrix;
    }
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            std::cout<<std::setprecision(3)<<(float)host_ptr[i*N + j]<<", ";
        }
        std::cout<<std::endl;
    }
    std::cout<<std::endl;
    if (device)
        delete[] host_ptr;
}

void clear_l2() 
{
    // Get actual L2 size via CUDA on first call of this function
    static int l2_clear_size = 0;
    static unsigned char* gpu_scratch_l2_clear = NULL;
    if (!gpu_scratch_l2_clear) {
        hipDeviceGetAttribute(&l2_clear_size, hipDeviceAttributeL2CacheSize, 0);
        l2_clear_size *= 2; // just to be extra safe (cache is not necessarily strict LRU)
        gpuErrchk(hipMalloc(&gpu_scratch_l2_clear, l2_clear_size));
    }
    // Clear L2 cache (this is run on every call unlike the above code)
    gpuErrchk(hipMemset(gpu_scratch_l2_clear, 0, l2_clear_size));
}

__global__ void matmul_elem(int n, datatype* a, datatype* b, datatype* c)
{
    int column = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    if (row < n && column < n)
    {
        float dot_prod = 0.f;
        for(int i = 0; i < n; i++)
        {
            dot_prod += (float)a[row*n + i] * (float)b[i*n + column];
        }
        c[row*n+column] = dot_prod;
    }
}

__global__ void tiled_matmul(int n, datatype* a, datatype* b, datatype* c)
{
    __shared__ datatype a_tile[TILE_WIDTH][TILE_WIDTH];
    __shared__ datatype b_tile[TILE_WIDTH][TILE_WIDTH];

    int column = blockIdx.x*TILE_WIDTH + threadIdx.x;
    int row = blockIdx.y*TILE_WIDTH + threadIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float dot_prod = 0.f;
    for (int tile_offset = 0; tile_offset<n; tile_offset+=TILE_WIDTH)
    {
        int a_chk = tile_offset+tx < n && row < n;
        a_tile[ty][tx] = a_chk ? a[row*n + tile_offset+tx] : (datatype)0.f;

        int b_chk = (tile_offset+ty) < n && column < n;
        b_tile[ty][tx] = b_chk ? b[(tile_offset+ty)*n + column] : (datatype)0.f;

        __syncthreads();
        for(int i = 0; i < TILE_WIDTH; i++)
        {
            dot_prod += (float)a_tile[ty][i] * (float)b_tile[i][tx];
        }
        __syncthreads();
    }

    if (row < n && column < n)
    {
        c[row*n+column] = dot_prod;
    }
}

using layout = nvcuda::wmma::row_major;
#define WMMA_MKN 16

__global__ void tensor_core_matmul(int n, datatype* a, datatype* b, datatype* c)
{
    const int32_t warpM = (blockIdx.x*blockDim.x+threadIdx.x)/32;
    const int32_t warpN = blockIdx.y*blockDim.y+threadIdx.y;

    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> a_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_MKN, WMMA_MKN, WMMA_MKN, half> acc;

    nvcuda::wmma::fill_fragment(acc, 0);

    for (int32_t i = 0; i < n; i+= WMMA_MKN)
    {
        const int32_t matrix_a_row = warpM * WMMA_MKN;
        const int32_t matrix_b_col = warpN * WMMA_MKN;

        if(matrix_a_row<n && matrix_b_col<n && i<n)
        {
            nvcuda::wmma::load_matrix_sync(a_frag, a + matrix_a_row * n + i, n);
            nvcuda::wmma::load_matrix_sync(b_frag, b + i * n + matrix_b_col, n);

            nvcuda::wmma::mma_sync(acc, a_frag, b_frag, acc);
        }
    }

    nvcuda::wmma::store_matrix_sync(c + warpM*WMMA_MKN*n + warpN*WMMA_MKN, acc, n, nvcuda::wmma::mem_row_major);
}

template<int WMMA_TILE_SIZE, int REG_TILES>
__global__ void tensor_core_matmul_smem(int n, datatype* a, datatype* b, datatype* c)
{
    const int32_t warpM = (blockIdx.x*blockDim.x+threadIdx.x)/32;
    const int32_t warpN = blockIdx.y*blockDim.y+threadIdx.y;
    __shared__ datatype smem[WMMA_TILE_SIZE*WMMA_MKN*WMMA_MKN];

    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> a_frag[REG_TILES];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> b_frag[REG_TILES];
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_MKN, WMMA_MKN, WMMA_MKN, half> acc;

    nvcuda::wmma::fill_fragment(acc, 0);
    const int32_t matrix_a_row = warpM * WMMA_MKN;

    for (int32_t tile = 0; tile < ceilf((float)n/(WMMA_MKN*WMMA_TILE_SIZE)); tile+=1)
    {
        for(int32_t i = threadIdx.x; i < WMMA_TILE_SIZE*WMMA_MKN*WMMA_MKN; i+=blockDim.x)
        {
            int32_t row = tile * WMMA_TILE_SIZE*WMMA_MKN + i/WMMA_MKN;
            int32_t column = warpN*WMMA_MKN + i%WMMA_MKN;
            if (row<n && column < n)
                smem[i] =  b[row*n + column];
        }
        __syncthreads();
        for (int32_t i = 0; i < WMMA_TILE_SIZE; i+=REG_TILES)
        {
            #pragma unroll REG_TILES
            for (int j = 0; j < REG_TILES; j++)
            {
                int idx = i + j;
                int mat_a_col = (tile*WMMA_TILE_SIZE+idx)*WMMA_MKN;
                if(matrix_a_row<n && mat_a_col<n)
                {
                    nvcuda::wmma::load_matrix_sync(a_frag[j], a + matrix_a_row * n + mat_a_col, n);
                    nvcuda::wmma::load_matrix_sync(b_frag[j], smem+idx*WMMA_MKN*WMMA_MKN, WMMA_MKN);

                    nvcuda::wmma::mma_sync(acc, a_frag[j], b_frag[j], acc);
                }
            }
        }
        __syncthreads();
    }

    if (warpM * WMMA_MKN < n && warpN*WMMA_MKN < n)
        nvcuda::wmma::store_matrix_sync(c + warpM*WMMA_MKN*n + warpN*WMMA_MKN, acc, n, nvcuda::wmma::mem_row_major);
}

template<int WMMA_TILE_SIZE, int OUT_TILES>
__global__ void tensor_core_matmul_smem2d(int n, datatype* a, datatype* b, datatype* c)
{
    const int32_t warpM = (blockIdx.x*blockDim.x+threadIdx.x)/32;
    const int32_t lane_id_x = threadIdx.x/32;
    const int32_t lane_id_y = threadIdx.y;
    const int32_t warpN = blockIdx.y*blockDim.y+threadIdx.y;
    __shared__ datatype a_smem[WMMA_TILE_SIZE][WMMA_TILE_SIZE][WMMA_MKN*WMMA_MKN];
    __shared__ datatype b_smem[WMMA_TILE_SIZE][WMMA_TILE_SIZE][WMMA_MKN*WMMA_MKN];

    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> a_frag[OUT_TILES][OUT_TILES];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> b_frag[OUT_TILES][OUT_TILES];
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_MKN, WMMA_MKN, WMMA_MKN, half> acc[OUT_TILES][OUT_TILES];

    for(int32_t i = 0; i<OUT_TILES; i++)
        for(int32_t j = 0; j<OUT_TILES; j++)
            nvcuda::wmma::fill_fragment(acc[i][j], 0);

    const int32_t matrix_a_row = warpM * WMMA_MKN * OUT_TILES;
    const int32_t matrix_b_col = warpN * WMMA_MKN * OUT_TILES;

    for (int32_t tile = 0; tile < ceilf((float)n/(WMMA_MKN*WMMA_TILE_SIZE)); tile+=1)
    {
        for(int32_t i = threadIdx.y*blockDim.x + threadIdx.x; i < WMMA_TILE_SIZE*WMMA_TILE_SIZE*WMMA_MKN*WMMA_MKN; i+=blockDim.x*blockDim.y)
        {
            int32_t tile_r = i/(WMMA_TILE_SIZE*WMMA_MKN*WMMA_MKN);
            int32_t tile_c = (i/(WMMA_MKN*WMMA_MKN))%WMMA_TILE_SIZE;
            int32_t tile_i = i%(WMMA_MKN*WMMA_MKN);
            int32_t row_a = blockIdx.x * WMMA_TILE_SIZE + tile_r*WMMA_MKN + tile_i/WMMA_MKN;
            int32_t column_a = tile * WMMA_TILE_SIZE*WMMA_MKN + tile_c * WMMA_MKN + tile_i%WMMA_MKN;
            if (row_a<n && column_a < n)
            {
                a_smem[tile_r][tile_c][tile_i] =  a[row_a*n + column_a];
            }
            int32_t row_b = tile * WMMA_TILE_SIZE*WMMA_MKN + tile_r*WMMA_MKN + tile_i/WMMA_MKN;
            int32_t column_b = blockIdx.y * WMMA_TILE_SIZE + tile_c * WMMA_MKN + tile_i%WMMA_MKN;
            if (row_b<n && column_b < n && tile_r == 1 && tile_c==1)
            if (row_b<n && column_b < n)
                b_smem[tile_r][tile_c][tile_i] =  b[row_b*n + column_b];
        }

        __syncthreads();
        for (int32_t i = 0; i < WMMA_TILE_SIZE; i+=OUT_TILES)
        {
            int32_t a_row = lane_id_x;
            int32_t b_col = lane_id_y;
            for (int col = 0; col < OUT_TILES; col++)
            {
                for (int row = 0; row < OUT_TILES; row++)
                {
                    nvcuda::wmma::load_matrix_sync(a_frag[row][col], a_smem[a_row + row][i + col], WMMA_MKN);
                    nvcuda::wmma::load_matrix_sync(b_frag[row][col], b_smem[i + row][b_col + col], WMMA_MKN);
                }
            }
            for (int col = 0; col < OUT_TILES; col++)
            {
                for (int row = 0; row < OUT_TILES; row++)
                {
                    for (int k = 0; k < OUT_TILES; k++)
                    {
                        nvcuda::wmma::mma_sync(acc[row][col], a_frag[row+k][col], b_frag[row][col+k], acc[row][col]);
                    }
                }
            }
        }
        __syncthreads();
    }

    for(int32_t i = 0; i<OUT_TILES; i++)
    {
        int32_t output_row = matrix_a_row + i*WMMA_MKN;
        for(int32_t j = 0; j<OUT_TILES; j++)
        {
            int32_t output_col = matrix_b_col + j*WMMA_MKN;
            if (output_row < n && output_col < n)
            {
                nvcuda::wmma::store_matrix_sync(c + output_row * n + output_col, acc[i][j], n, nvcuda::wmma::mem_row_major);
            }
        }
    }
}

void cpu_matmul(int n, datatype* a, datatype* b, datatype*c)
{
    for (int i = 0; i<n; i++)
    {
        for (int j = 0; j<n; j++)
        {
            datatype dot_product = 0.f;
            for (int k = 0; k<n; k++)
            {
                dot_product += a[i*n + k] * b[k*n + j];
            }
            c[i*n+j] = dot_product; 
        }
    }
}

    template <typename F>
double measure_performance(const F& fn)
{
    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
    double total_time = 0.0;

    for (int i = -WARMUP_STEPS; i<BENCH_STEPS; i++)
    {
        float run_time=0.0;
        clear_l2();
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipEventRecord(start));
        fn();
        gpuErrchk(hipEventRecord(stop));
        gpuErrchk(hipEventSynchronize(stop));
        gpuErrchk(hipEventElapsedTime(&run_time, start, stop));
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        if (i >= 0) // warmup
        {
            total_time += run_time;
        }
    }

    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));

    return total_time/BENCH_STEPS;
}

int main()
{
    std::vector<datatype*> outputs;
    float naive_times[TIMINGS];
    float tiled_times[TIMINGS];
    float cublas_times[TIMINGS];
    float tensor_core_times[TIMINGS];
    float tensor_core_smem_times[TIMINGS];
    float tensor_core_smem2d_times[TIMINGS];
    datatype* a_d;
    datatype* b_d;

    long max_N = std::pow<long, long>(2, START+TIMINGS-1);
    for(int i = 0; i < 6; i++)
    {
        datatype* output;
        hipMalloc((void**) &output, max_N*max_N*sizeof(datatype));
        hipMemset(output, 0, max_N*max_N*sizeof(datatype));
        outputs.push_back(output);
        
    }

    hipMalloc((void**) &a_d, max_N*max_N*sizeof(datatype));
    hipMalloc((void**) &b_d, max_N*max_N*sizeof(datatype));

    datatype* a = new datatype[max_N * max_N];
    datatype* b = new datatype[max_N * max_N];
    datatype* c = new datatype[max_N * max_N];

    std::random_device rd;
    std::mt19937 e2(rd());
    std::normal_distribution<> dist(-10, 10);

    for (int p = START; p<START+TIMINGS; p++)
    {
        long N = std::pow<long, long>(2, p);
        for (int i = 0; i<N; i++)
        {
            for (int j = 0; j<N; j++)
            {
                a[i*N + j] = 0;
                b[i*N + j] = dist(e2);
            }
            a[i*N + i] = dist(e2);
        }
        hipMemcpy(a_d, a, N*N*sizeof(datatype), hipMemcpyHostToDevice);
        hipMemcpy(b_d, b, N*N*sizeof(datatype), hipMemcpyHostToDevice);
        int BLOCK_SIZE=32;

        dim3 dimGrid(ceil(N/(float)BLOCK_SIZE), ceil(N/(float)BLOCK_SIZE), 1);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

        double matmul_time = measure_performance([&](){ matmul_elem<<<dimGrid, dimBlock>>>(N, a_d, b_d, outputs[0]); });


        dimGrid = dim3(ceil(N/(float)TILE_WIDTH), ceil(N/(float)TILE_WIDTH), 1);
        dimBlock = dim3(TILE_WIDTH, TILE_WIDTH, 1);

        double tiled_time = measure_performance([&](){ tiled_matmul<<<dimGrid, dimBlock>>>(N, a_d, b_d, outputs[1]); });

        hipblasHandle_t handle;
        hipblasCreate(&handle);
        datatype alpha = 1.f;
        datatype beta = 0.f;
        double cublas_time = measure_performance([&](){ hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N,N,N, &alpha, b_d, N, a_d, N, &beta, outputs[2], N); });
        ;
        int num_warps_x = 4;
        int num_warps_y = 4;
        dimBlock.x = num_warps_x * 32;
        dimBlock.y = num_warps_y;

        dimGrid.x = (N + (WMMA_MKN*num_warps_x -1)) / (WMMA_MKN*num_warps_x);
        dimGrid.y = (N + WMMA_MKN*num_warps_y -1) / (WMMA_MKN*num_warps_y);

        double tensor_cores_time = measure_performance([&](){ tensor_core_matmul<<<dimGrid, dimBlock>>>(N, a_d, b_d, outputs[3]); });

        num_warps_x = 32;
        num_warps_y = 1;
        dimBlock.x = num_warps_x * 32;
        dimBlock.y = num_warps_y;

        dimGrid.x = (N + (WMMA_MKN*num_warps_x -1)) / (WMMA_MKN*num_warps_x);
        dimGrid.y = (N + WMMA_MKN*num_warps_y -1) / (WMMA_MKN*num_warps_y);

        double tensor_cores_smem_time = measure_performance([&](){ tensor_core_matmul_smem<32, 8><<<dimGrid, dimBlock>>>(N, b_d, b_d, outputs[4]); });

        constexpr int SMEM_TILES = 2;
        constexpr int OUT_TILES = 2;

        num_warps_x = SMEM_TILES;
        num_warps_y = SMEM_TILES;
        dimBlock.x = num_warps_x * 32;
        dimBlock.y = num_warps_y;

        dimGrid.x = (N + (WMMA_MKN*num_warps_x -1)) / (WMMA_MKN*num_warps_x);
        dimGrid.y = (N + WMMA_MKN*num_warps_y -1) / (WMMA_MKN*num_warps_y);

        double _ = measure_performance([&](){ tensor_core_matmul_smem2d<SMEM_TILES, OUT_TILES><<<dimGrid, dimBlock>>>(N, a_d, b_d, outputs[4]); });
        double tensor_cores_smem2d_time = measure_performance([&](){ tensor_core_matmul_smem2d<SMEM_TILES, OUT_TILES><<<dimGrid, dimBlock>>>(N, a_d, b_d, outputs[5]); });

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        std::cout<<"n = "<<N<<" matmul time: "<<matmul_time<<
            " tiled time: "<<tiled_time<<
            " tensor cores time: "<<tensor_cores_time<<
            " tensor cores smem time: "<<tensor_cores_smem_time<<
            " tensor cores smem2d time: "<<tensor_cores_smem2d_time<<
            " cublas time: "<<cublas_time<<
            std::endl;


        naive_times[p-START] = matmul_time;
        tiled_times[p-START] = tiled_time;
        cublas_times[p-START] = cublas_time;
        tensor_core_times[p-START] = tensor_cores_time;
        tensor_core_smem_times[p-START] = tensor_cores_smem_time;
        tensor_core_smem2d_times[p-START] = tensor_cores_smem2d_time;
    }
    datatype* compare = new datatype[max_N*max_N];
    datatype* d_h = new datatype[max_N*max_N];
    hipMemcpy(compare, outputs[0], max_N*max_N*sizeof(datatype), hipMemcpyDeviceToHost);

    for(int i = 1; i < outputs.size(); i++)
    {
        hipMemcpy(d_h, outputs[i], max_N*max_N*sizeof(datatype), hipMemcpyDeviceToHost);
        float tolerance = 1e-8;
        for (int j = 0; j < max_N*max_N; j++)
        {
            ASSERT(abs((float)compare[j] - (float)d_h[j]) < tolerance, "failed at output %d, index %d, %f, %f\n", i, j, (float)d_h[j], (float)compare[j]);
        }
        hipFree(outputs[i]);
    }
    hipFree(a_d);
    hipFree(b_d);
    hipFree(compare);

    std::cout<<"normal_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<naive_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;

    std::cout<<"tiled_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<tiled_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;

    std::cout<<"cublas_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<cublas_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;

    std::cout<<"tensor_core_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<tensor_core_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;

    std::cout<<"tensor_core_smem_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<tensor_core_smem_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;

    std::cout<<"tensor_core_smem2d_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<tensor_core_smem2d_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;
    return 0;
}
