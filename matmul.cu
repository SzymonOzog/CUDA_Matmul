
#include <hip/hip_runtime.h>
#include <cmath>
#include <iomanip>
#include <iostream>
#include <cassert>
#include <hipblas.h>
#include <mma.h>
#include <random>
#include <vector>

#define TILE_WIDTH 32
#define BENCH_STEPS 100
#define WARMUP_STEPS 25
#define TIMINGS 2
#define START 11

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define ASSERT(cond, msg, args...) assert((cond) || !fprintf(stderr, (msg "\n"), args))

using datatype = half;
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void debug_print(datatype* matrix, int N, bool device)
{
    datatype* host_ptr;
    if (device)
    {
        host_ptr = new datatype[N*N];
        hipMemcpy(host_ptr, matrix, N*N*sizeof(datatype), hipMemcpyDeviceToHost);
    }
    else
    {
        host_ptr = matrix;
    }
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            std::cout<<std::setprecision(3)<<(float)host_ptr[i*N + j]<<", ";
        }
        std::cout<<std::endl;
    }
    std::cout<<std::endl;
    if (device)
        delete[] host_ptr;
}

void clear_l2() 
{
    // Get actual L2 size via CUDA on first call of this function
    static int l2_clear_size = 0;
    static unsigned char* gpu_scratch_l2_clear = NULL;
    if (!gpu_scratch_l2_clear) {
        hipDeviceGetAttribute(&l2_clear_size, hipDeviceAttributeL2CacheSize, 0);
        l2_clear_size *= 2; // just to be extra safe (cache is not necessarily strict LRU)
        gpuErrchk(hipMalloc(&gpu_scratch_l2_clear, l2_clear_size));
    }
    // Clear L2 cache (this is run on every call unlike the above code)
    gpuErrchk(hipMemset(gpu_scratch_l2_clear, 0, l2_clear_size));
}

__global__ void matmul_elem(int n, datatype* a, datatype* b, datatype* c)
{
    int column = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    if (row < n && column < n)
    {
        float dot_prod = 0.f;
        for(int i = 0; i < n; i++)
        {
            dot_prod += (float)a[row*n + i] * (float)b[i*n + column];
        }
        c[row*n+column] = dot_prod;
    }
}

__global__ void tiled_matmul(int n, datatype* a, datatype* b, datatype* c)
{
    __shared__ datatype a_tile[TILE_WIDTH][TILE_WIDTH];
    __shared__ datatype b_tile[TILE_WIDTH][TILE_WIDTH];

    int column = blockIdx.x*TILE_WIDTH + threadIdx.x;
    int row = blockIdx.y*TILE_WIDTH + threadIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float dot_prod = 0.f;
    for (int tile_offset = 0; tile_offset<n; tile_offset+=TILE_WIDTH)
    {
        int a_chk = tile_offset+tx < n && row < n;
        a_tile[ty][tx] = a_chk ? a[row*n + tile_offset+tx] : (datatype)0.f;

        int b_chk = (tile_offset+ty) < n && column < n;
        b_tile[ty][tx] = b_chk ? b[(tile_offset+ty)*n + column] : (datatype)0.f;

        __syncthreads();
        for(int i = 0; i < TILE_WIDTH; i++)
        {
            dot_prod += (float)a_tile[ty][i] * (float)b_tile[i][tx];
        }
        __syncthreads();
    }

    if (row < n && column < n)
    {
        c[row*n+column] = dot_prod;
    }
}

using layout = nvcuda::wmma::row_major;
#define WMMA_MKN 16

__global__ void tensor_core_matmul(int n, datatype* a, datatype* b, datatype* c)
{
    const int32_t warpM = (blockIdx.x*blockDim.x+threadIdx.x)/32;
    const int32_t warpN = blockIdx.y*blockDim.y+threadIdx.y;

    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> a_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_MKN, WMMA_MKN, WMMA_MKN, half> acc;

    nvcuda::wmma::fill_fragment(acc, 0);

    for (int32_t i = 0; i < n; i+= WMMA_MKN)
    {
        const int32_t matrix_a_row = warpM * WMMA_MKN;
        const int32_t matrix_b_col = warpN * WMMA_MKN;

        if(matrix_a_row<n && matrix_b_col<n && i<n)
        {
            nvcuda::wmma::load_matrix_sync(a_frag, a + matrix_a_row * n + i, n);
            nvcuda::wmma::load_matrix_sync(b_frag, b + i * n + matrix_b_col, n);

            nvcuda::wmma::mma_sync(acc, a_frag, b_frag, acc);
        }
    }

    nvcuda::wmma::store_matrix_sync(c + warpM*WMMA_MKN*n + warpN*WMMA_MKN, acc, n, nvcuda::wmma::mem_row_major);
}

template<int WMMA_TILE_SIZE, int REG_TILES>
__global__ void tensor_core_matmul_smem(int n, datatype* a, datatype* b, datatype* c)
{
    const int32_t warpM = (blockIdx.x*blockDim.x+threadIdx.x)/32;
    const int32_t warpN = blockIdx.y*blockDim.y+threadIdx.y;
    __shared__ datatype smem[WMMA_TILE_SIZE*WMMA_MKN*WMMA_MKN];

    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> a_frag[REG_TILES];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> b_frag[REG_TILES];
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_MKN, WMMA_MKN, WMMA_MKN, half> acc;

    nvcuda::wmma::fill_fragment(acc, 0);
    const int32_t matrix_a_row = warpM * WMMA_MKN;

    for (int32_t tile = 0; tile < ceilf((float)n/(WMMA_MKN*WMMA_TILE_SIZE)); tile+=1)
    {
        for(int32_t i = threadIdx.x; i < WMMA_TILE_SIZE*WMMA_MKN*WMMA_MKN; i+=blockDim.x)
        {
            int32_t row = tile * WMMA_TILE_SIZE*WMMA_MKN + i/WMMA_MKN;
            int32_t column = warpN*WMMA_MKN + i%WMMA_MKN;
            if (row<n && column < n)
                smem[i] =  b[row*n + column];
        }
        __syncthreads();
        for (int32_t i = 0; i < WMMA_TILE_SIZE; i+=REG_TILES)
        {
            #pragma unroll REG_TILES
            for (int j = 0; j < REG_TILES; j++)
            {
                int idx = i + j;
                int mat_a_col = (tile*WMMA_TILE_SIZE+idx)*WMMA_MKN;
                if(matrix_a_row<n && mat_a_col<n)
                {
                    nvcuda::wmma::load_matrix_sync(a_frag[j], a + matrix_a_row * n + mat_a_col, n);
                    nvcuda::wmma::load_matrix_sync(b_frag[j], smem+idx*WMMA_MKN*WMMA_MKN, WMMA_MKN);

                    nvcuda::wmma::mma_sync(acc, a_frag[j], b_frag[j], acc);
                }
            }
        }
        __syncthreads();
    }

    if (warpM * WMMA_MKN < n && warpN*WMMA_MKN < n)
        nvcuda::wmma::store_matrix_sync(c + warpM*WMMA_MKN*n + warpN*WMMA_MKN, acc, n, nvcuda::wmma::mem_row_major);
}


void cpu_matmul(int n, datatype* a, datatype* b, datatype*c)
{
    for (int i = 0; i<n; i++)
    {
        for (int j = 0; j<n; j++)
        {
            datatype dot_product = 0.f;
            for (int k = 0; k<n; k++)
            {
                dot_product += a[i*n + k] * b[k*n + j];
            }
            c[i*n+j] = dot_product; 
        }
    }
}

    template <typename F>
double measure_performance(const F& fn)
{
    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
    double total_time = 0.0;

    for (int i = -WARMUP_STEPS; i<BENCH_STEPS; i++)
    {
        float run_time=0.0;
        clear_l2();
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipEventRecord(start));
        fn();
        gpuErrchk(hipEventRecord(stop));
        gpuErrchk(hipEventSynchronize(stop));
        gpuErrchk(hipEventElapsedTime(&run_time, start, stop));
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        if (i >= 0) // warmup
        {
            total_time += run_time;
        }
    }

    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));

    return total_time/BENCH_STEPS;
}

int main()
{
    std::vector<datatype*> outputs;
    float naive_times[TIMINGS];
    float tiled_times[TIMINGS];
    float cublas_times[TIMINGS];
    float tensor_core_times[TIMINGS];
    float tensor_core_smem_times[TIMINGS];
    datatype* a_d;
    datatype* b_d;

    long max_N = std::pow<long, long>(2, START+TIMINGS-1);
    for(int i = 0; i < 5; i++)
    {
        datatype* output;
        hipMalloc((void**) &output, max_N*max_N*sizeof(datatype));
        hipMemset(output, 0, max_N*max_N*sizeof(datatype));
        outputs.push_back(output);
        
    }

    hipMalloc((void**) &a_d, max_N*max_N*sizeof(datatype));
    hipMalloc((void**) &b_d, max_N*max_N*sizeof(datatype));

    datatype* a = new datatype[max_N * max_N];
    datatype* b = new datatype[max_N * max_N];
    datatype* c = new datatype[max_N * max_N];

    std::random_device rd;
    std::mt19937 e2(rd());
    std::normal_distribution<> dist(-10, 10);

    for (int p = START; p<START+TIMINGS; p++)
    {
        long N = std::pow<long, long>(2, p);
        for (int i = 0; i<N; i++)
        {
            for (int j = 0; j<N; j++)
            {
                a[i*N + j] = 0;
                b[i*N + j] = dist(e2);
            }
            a[i*N + i] = dist(e2);
        }
        hipMemcpy(a_d, a, N*N*sizeof(datatype), hipMemcpyHostToDevice);
        hipMemcpy(b_d, b, N*N*sizeof(datatype), hipMemcpyHostToDevice);
        int BLOCK_SIZE=32;

        dim3 dimGrid(ceil(N/(float)BLOCK_SIZE), ceil(N/(float)BLOCK_SIZE), 1);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

        double matmul_time = measure_performance([&](){ matmul_elem<<<dimGrid, dimBlock>>>(N, a_d, b_d, outputs[0]); });


        dimGrid = dim3(ceil(N/(float)TILE_WIDTH), ceil(N/(float)TILE_WIDTH), 1);
        dimBlock = dim3(TILE_WIDTH, TILE_WIDTH, 1);

        double tiled_time = measure_performance([&](){ tiled_matmul<<<dimGrid, dimBlock>>>(N, a_d, b_d, outputs[1]); });

        hipblasHandle_t handle;
        hipblasCreate(&handle);
        datatype alpha = 1.f;
        datatype beta = 0.f;
        double cublas_time = measure_performance([&](){ hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N,N,N, &alpha, b_d, N, a_d, N, &beta, outputs[2], N); });
        ;
        int num_warps_x = 4;
        int num_warps_y = 4;
        dimBlock.x = num_warps_x * 32;
        dimBlock.y = num_warps_y;

        dimGrid.x = (N + (WMMA_MKN*num_warps_x -1)) / (WMMA_MKN*num_warps_x);
        dimGrid.y = (N + WMMA_MKN*num_warps_y -1) / (WMMA_MKN*num_warps_y);

        double tensor_cores_time = measure_performance([&](){ tensor_core_matmul<<<dimGrid, dimBlock>>>(N, a_d, b_d, outputs[3]); });

        num_warps_x = 32;
        num_warps_y = 1;
        dimBlock.x = num_warps_x * 32;
        dimBlock.y = num_warps_y;

        dimGrid.x = (N + (WMMA_MKN*num_warps_x -1)) / (WMMA_MKN*num_warps_x);
        dimGrid.y = (N + WMMA_MKN*num_warps_y -1) / (WMMA_MKN*num_warps_y);

        double tensor_cores_smem_time = measure_performance([&](){ tensor_core_matmul_smem<32, 8><<<dimGrid, dimBlock>>>(N, b_d, b_d, outputs[4]); });

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        std::cout<<"n = "<<N<<" matmul time: "<<matmul_time<<
            " tiled time: "<<tiled_time<<
            " tensor cores time: "<<tensor_cores_time<<
            " tensor cores smem time: "<<tensor_cores_smem_time<<
            " cublas time: "<<cublas_time<<
            std::endl;


        naive_times[p-START] = matmul_time;
        tiled_times[p-START] = tiled_time;
        cublas_times[p-START] = cublas_time;
        tensor_core_times[p-START] = tensor_cores_time;
        tensor_core_smem_times[p-START] = tensor_cores_smem_time;
    }
    datatype* compare = new datatype[max_N*max_N];
    datatype* d_h = new datatype[max_N*max_N];
    hipMemcpy(compare, outputs[0], max_N*max_N*sizeof(datatype), hipMemcpyDeviceToHost);
    for(int i = 1; i < outputs.size(); i++)
    {
        hipMemcpy(d_h, outputs[i], max_N*max_N*sizeof(datatype), hipMemcpyDeviceToHost);
        float tolerance = 1e-8;
        for (int j = 0; j < max_N*max_N; j++)
        {
            ASSERT(abs((float)compare[j] - (float)d_h[j]) < tolerance, "failed at output %d, index %d, %f, %f\n", i, j, (float)d_h[j], (float)compare[j]);
        }
        hipFree(outputs[i]);
    }
    hipFree(a_d);
    hipFree(b_d);
    hipFree(compare);

    std::cout<<"normal_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<naive_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;

    std::cout<<"tiled_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<tiled_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;

    std::cout<<"cublas_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<cublas_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;

    std::cout<<"tensor_core_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<tensor_core_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;

    std::cout<<"tensor_core_smem_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<tensor_core_smem_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;
    return 0;
}
