#include "hip/hip_runtime.h"
#include <cmath>
#include <iomanip>
#include <iostream>
#include <cassert>
#include <hipblas.h>
#include <mma.h>
#include <random>
#include <vector>

#define TILE_WIDTH 32
#define BENCH_STEPS 1
#define WARMUP_STEPS 0
#define TIMINGS 1
#define START 6
#define WMMA_MKN 16

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
#define ASSERT(cond, msg, args...) assert((cond) || !fprintf(stderr, (msg "\n"), args))

using datatype = half;
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

void debug_print(datatype* matrix, int N, bool device)
{
    datatype* host_ptr;
    if (device)
    {
        host_ptr = new datatype[N*N];
        hipMemcpy(host_ptr, matrix, N*N*sizeof(datatype), hipMemcpyDeviceToHost);
    }
    else
    {
        host_ptr = matrix;
    }

    const int col_width = 8;

    std::cout << std::endl;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            std::cout << std::setw(col_width)
                      << std::fixed << std::setprecision(3)
                      << static_cast<float>(host_ptr[i*N + j]) << " ";

            if (j % WMMA_MKN == WMMA_MKN - 1)
                std::cout << " | ";
        }
        std::cout << std::endl;

        if (i % WMMA_MKN == WMMA_MKN - 1)
        {
            std::cout << std::string(N * (col_width + 1) + (N / WMMA_MKN) * 3, '_') << std::endl;
        }
    }
    std::cout << std::endl;
    std::cout << std::endl;

    if (device)
        delete[] host_ptr;
}

void clear_l2() 
{
    // Get actual L2 size via CUDA on first call of this function
    static int l2_clear_size = 0;
    static unsigned char* gpu_scratch_l2_clear = NULL;
    if (!gpu_scratch_l2_clear) {
        hipDeviceGetAttribute(&l2_clear_size, hipDeviceAttributeL2CacheSize, 0);
        l2_clear_size *= 2; // just to be extra safe (cache is not necessarily strict LRU)
        gpuErrchk(hipMalloc(&gpu_scratch_l2_clear, l2_clear_size));
    }
    // Clear L2 cache (this is run on every call unlike the above code)
    gpuErrchk(hipMemset(gpu_scratch_l2_clear, 0, l2_clear_size));
}

__global__ void matmul_elem(int n, datatype* a, datatype* b, datatype* c)
{
    int column = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    if (row < n && column < n)
    {
        float dot_prod = 0.f;
        for(int i = 0; i < n; i++)
        {
            dot_prod += (float)a[row*n + i] * (float)b[i*n + column];
        }
        c[row*n+column] = dot_prod;
    }
}

__global__ void tiled_matmul(int n, datatype* a, datatype* b, datatype* c)
{
    __shared__ datatype a_tile[TILE_WIDTH][TILE_WIDTH];
    __shared__ datatype b_tile[TILE_WIDTH][TILE_WIDTH];

    int column = blockIdx.x*TILE_WIDTH + threadIdx.x;
    int row = blockIdx.y*TILE_WIDTH + threadIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float dot_prod = 0.f;
    for (int tile_offset = 0; tile_offset<n; tile_offset+=TILE_WIDTH)
    {
        int a_chk = tile_offset+tx < n && row < n;
        a_tile[ty][tx] = a_chk ? a[row*n + tile_offset+tx] : (datatype)0.f;

        int b_chk = (tile_offset+ty) < n && column < n;
        b_tile[ty][tx] = b_chk ? b[(tile_offset+ty)*n + column] : (datatype)0.f;

        __syncthreads();
        for(int i = 0; i < TILE_WIDTH; i++)
        {
            dot_prod += (float)a_tile[ty][i] * (float)b_tile[i][tx];
        }
        __syncthreads();
    }

    if (row < n && column < n)
    {
        c[row*n+column] = dot_prod;
    }
}

using layout = nvcuda::wmma::row_major;

__global__ void tensor_core_matmul(int n, datatype* a, datatype* b, datatype* c)
{
    const int32_t warpM = (blockIdx.x*blockDim.x+threadIdx.x)/32;
    const int32_t warpN = blockIdx.y*blockDim.y+threadIdx.y;

    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> a_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_MKN, WMMA_MKN, WMMA_MKN, half> acc;

    nvcuda::wmma::fill_fragment(acc, 0);

    for (int32_t i = 0; i < n; i+= WMMA_MKN)
    {
        const int32_t matrix_a_row = warpM * WMMA_MKN;
        const int32_t matrix_b_col = warpN * WMMA_MKN;

        if(matrix_a_row<n && matrix_b_col<n && i<n)
        {
            nvcuda::wmma::load_matrix_sync(a_frag, a + matrix_a_row * n + i, n);
            nvcuda::wmma::load_matrix_sync(b_frag, b + i * n + matrix_b_col, n);

            nvcuda::wmma::mma_sync(acc, a_frag, b_frag, acc);
        }
    }

    nvcuda::wmma::store_matrix_sync(c + warpM*WMMA_MKN*n + warpN*WMMA_MKN, acc, n, nvcuda::wmma::mem_row_major);
}

template<int WMMA_TILE_SIZE, int OUT_TILES>
__global__ void tensor_core_matmul_smem2d(int n, datatype* a, datatype* b, datatype* c)
{
    const int32_t warpM = (blockIdx.x*blockDim.x+threadIdx.x)/32;
    const int32_t lane_id_x = threadIdx.x/32;
    const int32_t lane_id_y = threadIdx.y;
    const int32_t warpN = blockIdx.y*blockDim.y+threadIdx.y;
    
    extern __shared__ char smem[];

    datatype (*a_smem)[WMMA_TILE_SIZE][WMMA_MKN*WMMA_MKN]
        = reinterpret_cast<datatype(*)[WMMA_TILE_SIZE][WMMA_MKN*WMMA_MKN]>(smem);
    datatype (*b_smem)[WMMA_TILE_SIZE][WMMA_MKN*WMMA_MKN]
        = reinterpret_cast<datatype(*)[WMMA_TILE_SIZE][WMMA_MKN*WMMA_MKN]>(
                smem + WMMA_TILE_SIZE*WMMA_TILE_SIZE*WMMA_MKN*WMMA_MKN*sizeof(datatype));

    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> a_frag[OUT_TILES][OUT_TILES];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> b_frag[OUT_TILES][OUT_TILES];
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_MKN, WMMA_MKN, WMMA_MKN, half> acc[OUT_TILES][OUT_TILES];

    for(int32_t i = 0; i<OUT_TILES; i++)
        for(int32_t j = 0; j<OUT_TILES; j++)
            nvcuda::wmma::fill_fragment(acc[i][j], 0);

    const int32_t matrix_a_row = warpM * WMMA_MKN * OUT_TILES;
    const int32_t matrix_b_col = warpN * WMMA_MKN * OUT_TILES;

    for (int32_t tile = 0; tile < ceilf((float)n/(WMMA_MKN*WMMA_TILE_SIZE)); tile+=1)
    {
        for(int32_t i = threadIdx.y*blockDim.x + threadIdx.x; i < WMMA_TILE_SIZE*WMMA_TILE_SIZE*WMMA_MKN*WMMA_MKN; i+=blockDim.x*blockDim.y)
        {
            int32_t tile_r = i/(WMMA_TILE_SIZE*WMMA_MKN*WMMA_MKN);
            int32_t tile_c = (i/(WMMA_MKN*WMMA_MKN))%WMMA_TILE_SIZE;
            int32_t tile_i = i%(WMMA_MKN*WMMA_MKN);
            int32_t row_a = blockIdx.x * WMMA_TILE_SIZE*WMMA_MKN + tile_r*WMMA_MKN + tile_i/WMMA_MKN;
            int32_t column_a = tile * WMMA_TILE_SIZE*WMMA_MKN + tile_c * WMMA_MKN + tile_i%WMMA_MKN;
            if (row_a<n && column_a < n)
            {
                a_smem[tile_r][tile_c][tile_i] =  a[row_a*n + column_a];
            }
            int32_t row_b = tile * WMMA_TILE_SIZE*WMMA_MKN + tile_r*WMMA_MKN + tile_i/WMMA_MKN;
            int32_t column_b = blockIdx.y * WMMA_TILE_SIZE*WMMA_MKN + tile_c * WMMA_MKN + tile_i%WMMA_MKN;
            if (row_b<n && column_b < n)
                b_smem[tile_r][tile_c][tile_i] =  b[row_b*n + column_b];
        }

        __syncthreads();
        for (int32_t i = 0; i < WMMA_TILE_SIZE; i+=OUT_TILES)
        {
            int32_t a_row = lane_id_x*OUT_TILES;
            int32_t b_col = lane_id_y*OUT_TILES;
            for (int col = 0; col < OUT_TILES; col++)
            {
                for (int row = 0; row < OUT_TILES; row++)
                {
                    nvcuda::wmma::load_matrix_sync(a_frag[row][col], a_smem[a_row + row][i + col], WMMA_MKN);
                    nvcuda::wmma::load_matrix_sync(b_frag[row][col], b_smem[i + row][b_col + col], WMMA_MKN);
                }
            }
            for (int col = 0; col < OUT_TILES; col++)
            {
                for (int row = 0; row < OUT_TILES; row++)
                {
                    for (int k = 0; k < OUT_TILES; k++)
                    {
                        nvcuda::wmma::mma_sync(acc[row][col], a_frag[row][k], b_frag[k][col], acc[row][col]);
                    }
                }
            }
        }
        __syncthreads();
    }

    for(int32_t i = 0; i<OUT_TILES; i++)
    {
        int32_t output_row = matrix_a_row + i*WMMA_MKN;
        for(int32_t j = 0; j<OUT_TILES; j++)
        {
            int32_t output_col = matrix_b_col + j*WMMA_MKN;
            if (output_row < n && output_col < n)
            {
                nvcuda::wmma::store_matrix_sync(c + output_row * n + output_col, acc[i][j], n, nvcuda::wmma::mem_row_major);
            }
        }
    }
}

template<int OUT_TILES>
__global__ void tensor_core_matmul_reg(int n, datatype* a, datatype* b, datatype* c)
{
    const int32_t warpM = (blockIdx.x*blockDim.x+threadIdx.x)/32;
    const int32_t warpN = blockIdx.y*blockDim.y+threadIdx.y;

    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> a_frag[OUT_TILES];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_MKN, WMMA_MKN, WMMA_MKN, half, layout> b_frag;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_MKN, WMMA_MKN, WMMA_MKN, half> acc[OUT_TILES][OUT_TILES];

    for(int32_t i = 0; i<OUT_TILES; i++)
        for(int32_t j = 0; j<OUT_TILES; j++)
            nvcuda::wmma::fill_fragment(acc[i][j], 0);

    const int32_t matrix_a_row = warpM * WMMA_MKN * OUT_TILES;
    const int32_t matrix_b_col = warpN * WMMA_MKN * OUT_TILES;

    for (int32_t tile = 0; tile < n; tile+=OUT_TILES*WMMA_MKN)
    {
        for (int out_col = 0; out_col < OUT_TILES; out_col++)
        {
            for (int out_row = 0; out_row < OUT_TILES; out_row++)
            {
                int32_t a_row = matrix_a_row + out_row*WMMA_MKN;
                int32_t a_col = tile + out_col*WMMA_MKN;
                if(a_row < n && a_col < n)
                {
                    nvcuda::wmma::load_matrix_sync(a_frag[out_row], a + a_row*n + a_col, n);
                }
            }
            for (int out_row = 0; out_row < OUT_TILES; out_row++)
            {
                int32_t b_col = matrix_b_col + (out_row)*WMMA_MKN;
                int32_t b_row = tile + out_col*WMMA_MKN;
                nvcuda::wmma::load_matrix_sync(b_frag, b + b_row*n + b_col, n);
                for (int k = 0; k < OUT_TILES && tile + k*WMMA_MKN < n; k++)
                {
                    nvcuda::wmma::mma_sync(acc[k][out_row], a_frag[k], b_frag, acc[k][out_row]);
                }
            }
        }
    }

    for(int32_t i = 0; i<OUT_TILES; i++)
    {
        int32_t output_row = matrix_a_row + i*WMMA_MKN;
        for(int32_t j = 0; j<OUT_TILES; j++)
        {
            int32_t output_col = matrix_b_col + j*WMMA_MKN;
            if (output_row < n && output_col < n)
            {
                nvcuda::wmma::store_matrix_sync(c + output_row * n + output_col, acc[i][j], n, nvcuda::wmma::mem_row_major);
            }
        }
    }
}

void cpu_matmul(int n, datatype* a, datatype* b, datatype*c)
{
    for (int i = 0; i<n; i++)
    {
        for (int j = 0; j<n; j++)
        {
            datatype dot_product = 0.f;
            for (int k = 0; k<n; k++)
            {
                dot_product += a[i*n + k] * b[k*n + j];
            }
            c[i*n+j] = dot_product; 
        }
    }
}

    template <typename F>
double measure_performance(const F& fn)
{
    hipEvent_t start, stop;
    gpuErrchk(hipEventCreate(&start));
    gpuErrchk(hipEventCreate(&stop));
    double total_time = 0.0;

    for (int i = -WARMUP_STEPS; i<BENCH_STEPS; i++)
    {
        float run_time=0.0;
        clear_l2();
        gpuErrchk(hipDeviceSynchronize());
        gpuErrchk(hipEventRecord(start));
        fn();
        gpuErrchk(hipEventRecord(stop));
        gpuErrchk(hipEventSynchronize(stop));
        gpuErrchk(hipEventElapsedTime(&run_time, start, stop));
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        if (i >= 0) // warmup
        {
            total_time += run_time;
        }
    }

    gpuErrchk(hipEventDestroy(start));
    gpuErrchk(hipEventDestroy(stop));

    return total_time/BENCH_STEPS;
}

int main()
{
    std::vector<datatype*> outputs;
    float naive_times[TIMINGS];
    float tiled_times[TIMINGS];
    float cublas_times[TIMINGS];
    float tensor_core_times[TIMINGS];
    float tensor_core_smem2d_times[TIMINGS];
    datatype* a_d;
    datatype* b_d;

    long max_N = std::pow<long, long>(2, START+TIMINGS-1);
    for(int i = 0; i < 5; i++)
    {
        datatype* output;
        hipMalloc((void**) &output, max_N*max_N*sizeof(datatype));
        hipMemset(output, 0, max_N*max_N*sizeof(datatype));
        outputs.push_back(output);
        
    }

    hipMalloc((void**) &a_d, max_N*max_N*sizeof(datatype));
    hipMalloc((void**) &b_d, max_N*max_N*sizeof(datatype));

    datatype* a = new datatype[max_N * max_N];
    datatype* b = new datatype[max_N * max_N];
    datatype* c = new datatype[max_N * max_N];

    std::random_device rd;
    std::mt19937 e2(rd());
    std::normal_distribution<> dist(0, 2);

    for (int p = START; p<START+TIMINGS; p++)
    {
        long N = std::pow<long, long>(2, p);
        for (int i = 0; i<N; i++)
        {
            for (int j = 0; j<N; j++)
            {
                a[i*N + j] = 0;
                b[i*N + j] = dist(e2);
            }
            a[i*N + i] = dist(e2);
        }
        hipMemcpy(a_d, a, N*N*sizeof(datatype), hipMemcpyHostToDevice);
        hipMemcpy(b_d, b, N*N*sizeof(datatype), hipMemcpyHostToDevice);
        int BLOCK_SIZE=32;

        dim3 dimGrid(ceil(N/(float)BLOCK_SIZE), ceil(N/(float)BLOCK_SIZE), 1);
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);

        double matmul_time = measure_performance([&](){ matmul_elem<<<dimGrid, dimBlock>>>(N, a_d, b_d, outputs[0]); });


        dimGrid = dim3(ceil(N/(float)TILE_WIDTH), ceil(N/(float)TILE_WIDTH), 1);
        dimBlock = dim3(TILE_WIDTH, TILE_WIDTH, 1);

        double tiled_time = measure_performance([&](){ tiled_matmul<<<dimGrid, dimBlock>>>(N, a_d, b_d, outputs[1]); });

        hipblasHandle_t handle;
        hipblasCreate(&handle);
        datatype alpha = 1.f;
        datatype beta = 0.f;
        double cublas_time = measure_performance([&](){ hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N,N,N, &alpha, b_d, N, a_d, N, &beta, outputs[2], N); });

        int num_warps_x = 4;
        int num_warps_y = 4;
        dimBlock.x = num_warps_x * 32;
        dimBlock.y = num_warps_y;

        dimGrid.x = (N + (WMMA_MKN*num_warps_x -1)) / (WMMA_MKN*num_warps_x);
        dimGrid.y = (N + WMMA_MKN*num_warps_y -1) / (WMMA_MKN*num_warps_y);

        double tensor_cores_time = measure_performance([&](){ tensor_core_matmul<<<dimGrid, dimBlock>>>(N, a_d, b_d, outputs[3]); });

        constexpr int SMEM_TILES = 4;
        constexpr int OUT_TILES = 1;

        num_warps_x = SMEM_TILES/OUT_TILES;
        num_warps_y = SMEM_TILES/OUT_TILES;
        dimBlock.x = num_warps_x * 32;
        dimBlock.y = num_warps_y;

        dimGrid.x = std::ceil((float)N/(SMEM_TILES*WMMA_MKN));
        dimGrid.y = std::ceil((float)N/(SMEM_TILES*WMMA_MKN));
        unsigned int smem_size = 2*SMEM_TILES*SMEM_TILES*WMMA_MKN*WMMA_MKN*sizeof(datatype);
        hipFuncSetAttribute(reinterpret_cast<const void*>(tensor_core_matmul_smem2d<SMEM_TILES), OUT_TILES>, hipFuncAttributeMaxDynamicSharedMemorySize, smem_size);

        double tensor_cores_smem_time = measure_performance([&](){ tensor_core_matmul_smem2d<SMEM_TILES, OUT_TILES><<<dimGrid, dimBlock, smem_size>>>(N, a_d, b_d, outputs[4]); });

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        constexpr int OUT_TILES2 = 2;
        num_warps_x = 4;
        num_warps_y = 4;
        dimBlock.x = num_warps_x * 32;
        dimBlock.y = num_warps_y;

        dimGrid.x = std::ceil((float)N/(OUT_TILES2*WMMA_MKN*num_warps_x));
        dimGrid.y = std::ceil((float)N/(OUT_TILES2*WMMA_MKN*num_warps_y));
        double tensor_cores_reg_time = measure_performance([&](){ tensor_core_matmul_reg<OUT_TILES2><<<dimGrid, dimBlock>>>(N, a_d, b_d, outputs[5]); });

        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());
        long ops = 2*std::pow(N, 3);
        std::cout<<"n = "<<N<<" matmul time: "<<matmul_time<< " gflops: " <<(double)ops/(matmul_time*1e6) <<
            "\n tiled time: "<<tiled_time<< " gflops: " <<(double)ops/(tiled_time*1e6) <<
            "\n tensor cores time: "<<tensor_cores_time<< " gflops: " <<(double)ops/(tensor_cores_time*1e6) <<
            "\n tensor cores smem2d time: "<<tensor_cores_smem_time<< " gflops: " <<(double)ops/(tensor_cores_smem_time*1e6) <<
            "\n tensor cores reg time: "<<tensor_cores_reg_time<< " gflops: " <<(double)ops/(tensor_cores_reg_time*1e6) <<
            "\n cublas time: "<<cublas_time<< " gflops: " <<(double)ops/(cublas_time*1e6) <<
            "\n -------------------------------------------------------------------------------------" <<
            std::endl;

        naive_times[p-START] = matmul_time;
        tiled_times[p-START] = tiled_time;
        cublas_times[p-START] = cublas_time;
        tensor_core_times[p-START] = tensor_cores_time;
        tensor_core_smem2d_times[p-START] = tensor_cores_smem_time;
        tensor_core_reg_times[p-START] = tensor_cores_reg_time;
    }
    datatype* compare = new datatype[max_N*max_N];
    datatype* d_h = new datatype[max_N*max_N];
    hipMemcpy(compare, outputs[3], max_N*max_N*sizeof(datatype), hipMemcpyDeviceToHost);

    for(int i = 3; i < outputs.size(); i++)
    {
        hipMemcpy(d_h, outputs[i], max_N*max_N*sizeof(datatype), hipMemcpyDeviceToHost);
        float tolerance = 1e-3;
        for (int j = 0; j < max_N*max_N; j++)
        {
            float relative_difference = abs(1 - ((float)compare[j] / (float)d_h[j]));
            ASSERT(relative_difference < tolerance, "failed at output %d, index %d, %f, %f, rdiff; %f\n", i, j, (float)d_h[j], (float)compare[j], relative_difference);
        } 
        hipFree(outputs[i]);
    }
    hipFree(a_d);
    hipFree(b_d);
    hipFree(compare);

    std::cout<<"normal_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<naive_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;

    std::cout<<"tiled_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<tiled_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;

    std::cout<<"cublas_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<cublas_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;

    std::cout<<"tensor_core_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<tensor_core_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;

    std::cout<<"tensor_core_smem2d_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<tensor_core_smem2d_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;

    std::cout<<"tensor_core_reg_times = [";
    for (int i = 0; i<TIMINGS; i++)
    {
        std::cout<<tensor_core_reg_times[i]<<", ";
    }
    std::cout<<"]"<<std::endl;
    return 0;
}
